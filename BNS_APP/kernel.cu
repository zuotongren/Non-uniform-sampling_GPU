#include "hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include "hip/device_functions.h"
__constant__ double dev_a[Samp_point];
__global__ void addKernel(double *PSD_sample,double * seque)
{
	
	const int i=threadIdx.x+blockIdx.x*blockDim.x;
	const double k=0.490*(pow(0.1, -(5.0 / 3.0)));
	PSD_sample[i] = k*pow((pow(seque[blockIdx.x],2.0)+pow(seque[threadIdx.x],2.0)),-(11.0/6.0));
}


__global__ void Kernel(double_complex*Phase , double_complex * Rand_Spect_Lin ,int sm,int D)
{
	__shared__ double_complex Phase_part[Samp_point];//����һ�������ڴ棬���ڴ�����ɵĽ��
	const double DX=1.0*D/POINT;
	int s=0;
	const int block_Idx = gridDim.x * blockIdx.y + blockIdx.x;
	double sum1=0.0,sum2=0.0,sum3=0.0,sum4=0.0;
	double x=dev_a[blockIdx.x]*(( sm + 1)*DX)+dev_a[threadIdx.x]*((blockIdx.y +1)*DX);
	if(blockIdx.x==Samp_point-1 || threadIdx.x == Samp_point-1 )
	{
		Phase_part[threadIdx.x].real = 0.0;
		Phase_part[threadIdx.x].imag = 0.0;
	}
	else
	{
      sum1=Rand_Spect_Lin[( blockDim.x )*blockIdx.x+threadIdx.x].real*(cos(x));
	  sum2=Rand_Spect_Lin[( blockDim.x )*blockIdx.x+threadIdx.x].imag*(sin(x));
	  sum3=Rand_Spect_Lin[( blockDim.x )*blockIdx.x+threadIdx.x].real*(sin(x));
	  sum4=Rand_Spect_Lin[( blockDim.x )*blockIdx.x+threadIdx.x].imag*(cos(x));
	  Phase_part[threadIdx.x].real = sum1 - sum2;
	  Phase_part[threadIdx.x].imag = sum3 + sum4;
	}
	__syncthreads();
	int j=blockDim.x/2;
	while(j!=0)
	{
	   if(s<=threadIdx.x && threadIdx.x< j)
		{
			Phase_part[threadIdx.x ].real += Phase_part[threadIdx.x - s + j].real;
			Phase_part[threadIdx.x ].imag += Phase_part[threadIdx.x - s + j].imag;
		}
		__syncthreads();
	   if(j%2!=0 && j!=1)
		{
			j=j+1;
			s=1;
		}
	   else s=0;
		j /=2;
	}
	if(threadIdx.x == 0)
		Phase[block_Idx]=Phase_part[0];


}
//__constant__ double dev_a[257];
// Helper function for using CUDA to add vectors in parallel.
int cuda_PSD_sample(int Nf,double *PSD_sample,double *sequex)
{
	double *dev_a = 0;
    double *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
	// Allocate GPU buffers for three vectors (one input, one output).
    cudaStatus = hipMalloc((void**)&dev_a, Nf * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_c, Nf*Nf * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
	cudaStatus = hipMemcpy(dev_a,sequex, Nf * sizeof(double),hipMemcpyHostToDevice);//, hipMemcpyHostToDevice
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
     //Copy input vectors from host memory to GPU buffers.
	//cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(dev_a),sequex, Nf * sizeof(double));//, hipMemcpyHostToDevice
 //   if (cudaStatus != hipSuccess) {
 //       fprintf(stderr, "hipMemcpy failed!");
 //       goto Error;
 //   }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<Nf,Nf>>>(dev_c,dev_a);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
	
		 cudaStatus = hipMemcpy(PSD_sample, dev_c, Nf*Nf*sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
	}
   
Error:
	hipFree(dev_a);
	hipFree(dev_c);
    
    return cudaStatus;
}
/*************************************************************************/
//
//
//
//
//
//
//
//
//
//***************************************************************************/
hipError_t cuda_Phase(double_complex*Phase,double_complex * Rand_Spect_Lin,double*ka_dbside,int sm,int D)
{
	int bbb=11;
	dim3 blocks(Samp_point,POINT);
	int threadsPerBlock=Samp_point;
    double_complex *dev_c = 0;               //��GPU�������� ���л����� ���ڴ����λ��������
	double_complex *dev_b = 0;      //��GPU�������� ���л����� ���ڴ�Ź���������������֮���ֵ
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
		bbb=1;
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
	// Allocate GPU buffers for three vectors (one input, one output).
	 cudaStatus = hipMalloc((void**)&dev_b, Samp_point*Samp_point* sizeof(double_complex));
    if (cudaStatus != hipSuccess) {
		bbb=2;
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_c, Samp_point*POINT* sizeof(double_complex));
    if (cudaStatus != hipSuccess) {
		bbb=3;
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
     //Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(dev_a),ka_dbside, Samp_point * sizeof(double));//, hipMemcpyHostToDevice
    if (cudaStatus != hipSuccess) {
		bbb=5;
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	cudaStatus = hipMemcpy(dev_b,Rand_Spect_Lin,Samp_point*Samp_point*sizeof(double_complex), hipMemcpyHostToDevice);//, hipMemcpyHostToDevice
    if (cudaStatus != hipSuccess) {
		bbb=7;
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    Kernel<<<blocks,threadsPerBlock>>>(dev_c,dev_b,sm,D);

    //Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
		bbb=8;
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
		bbb=9;
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
	
		 cudaStatus = hipMemcpy(Phase, dev_c,Samp_point*POINT*sizeof(double_complex), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
		bbb=10;
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
	}
   
Error:
	  //hipFree(dev_a);
	  hipFree(dev_b);
	  hipFree(dev_c);
      return cudaStatus;
}
